#include "hip/hip_runtime.h"
/**
 * @brief JUDAH - Jacob is equipped with a text-based user interface
 *
 * @file qlc.cu
 * @author  Norbert Bátfai <nbatfai@gmail.com>
 * @version 0.0.1
 *
 * @section LICENSE
 *
 * Copyright (C) 2015 Norbert Bátfai, batfai.norbert@inf.unideb.hu
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 *
 * @section DESCRIPTION
 *
 * JACOB, https://github.com/nbatfai/jacob
 *
 * "The son of Isaac is Jacob." The project called Jacob is an experiment 
 * to replace Isaac's (GUI based) visual imagination with a character console.
 *
 * ISAAC, https://github.com/nbatfai/isaac
 *
 * "The son of Samu is Isaac." The project called Isaac is a case study 
 * of using deep Q learning with neural networks for predicting the next 
 * sentence of a conversation.
 * 
 * SAMU, https://github.com/nbatfai/samu
 *
 * The main purpose of this project is to allow the evaluation and 
 * verification of the results of the paper entitled "A disembodied 
 * developmental robotic agent called Samu Bátfai". It is our hope 
 * that Samu will be the ancestor of developmental robotics chatter 
 * bots that will be able to chat in natural language like humans do.
 *
 */

#include "qlc.h"

__device__ double
sigmoid ( double x )
{
    return 1.0/ ( 1.0 + exp ( -x ) );
}

__device__ double
prcp ( int j, int nu, double *newu, double *u, double *w )
{
    newu[j] = 0.0;
    for ( int k = 0; k < nu; ++k ) {
        newu[j] += w[j*nu+k] * u[k];
    }
    return sigmoid ( newu[j] );
}

__global__ void
layer_kernel ( int nu, double *newu, double *u, double *w )
{
    //int j = blockIdx.x;
    int j = threadIdx.x;
    newu[j] = prcp ( j, nu, newu, u, w );
}

void cuda_layer ( int i, int* n_units,   double **units,   double ***weights )
{
    double *device_newu;
    hipMalloc ( ( void ** ) &device_newu, n_units[i] * sizeof ( double ) );

    double *device_u;
    hipMalloc ( ( void ** ) &device_u, n_units[i-1] * sizeof ( double ) );
    hipMemcpy ( device_u, units[i-1],
                 n_units[i-1]*sizeof ( double ), hipMemcpyHostToDevice );

    double *device_w;
    hipMalloc ( ( void ** ) &device_w, n_units[i] * n_units[i-1] * sizeof ( double ) );
    for ( int wi = 0; wi<n_units[i]; ++wi ) {

        hipMemcpy ( device_w+wi*n_units[i-1], weights[i-1][wi],
                     n_units[i-1]*sizeof ( double ), hipMemcpyHostToDevice );
    }
    ///*
    dim3 grid ( 1, 1 );
    dim3 tgrid ( n_units[i] , 1 );
    layer_kernel <<< grid, tgrid >>> ( n_units[i-1], device_newu, device_u, device_w );
    //*/
    /*
    dim3 grid ( n_units[i] , 1 );
    layer_kernel <<< grid, 1 >>> ( n_units[i-1], device_newu, device_u, device_w );
    */
    hipMemcpy ( units[i], device_newu,
                 n_units[i]*sizeof ( double ), hipMemcpyDeviceToHost );

    hipFree ( device_newu );
    hipFree ( device_u );
    hipFree ( device_w );

}
